#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include<sys/wait.h> 
#include<unistd.h> 

__global__ void matrixVectorMultCuda(int* row_ptr, int* col_ind, float* values, float* x,int row_size , int col_size, int value_size ,int threadSize);
__host__ void outer_VecMatMult_Started (struct SparseMatrix* sparse_Matrix, float *x, int row_size, int col_size, int value_size, int NUMBER_OF_CUDA_THREADS ,int NUMBER_OF_REPETITION , int OUTPUT_FLAG);

struct SparseMatrix{
    int *row_ptr;
    int *col_ind;
    float *values;
};

struct SparseMatrix* readSparseMatrix (char* arg, int &row_size, int &col_size, int &value_size){
    FILE * file;
    int i,j, int_tmp;
    float float_tmp;
    struct SparseMatrix *matrix =  (struct SparseMatrix *) malloc(sizeof(struct SparseMatrix));  
        
    if ((file = fopen(arg, "r+")) == NULL)
    {
        printf("ERROR: file open failed\n");
        //return(NULL);
    }
    for(int k = 0; k<3; k++){
        fscanf(file,"%d", &int_tmp);
        switch( k )
        {
            case 0:
                row_size = int_tmp;
               
                break;
            case 1:
                col_size = int_tmp;
                
                break;
            case 2:
                value_size = int_tmp;
               
                break;   
        }
    }
    matrix->row_ptr = (int*) malloc(sizeof(int)*value_size);
    matrix->col_ind = (int*) malloc(sizeof(int)*value_size);
    matrix->values =  (float*) malloc(sizeof(float)*value_size);                     
    for (i = 0; i < (value_size); i++){
        for (j = 0; j < (3); j++){
            switch( j ){
                case 0:
                    fscanf(file,"%d", &int_tmp);
                    matrix->row_ptr[i] = int_tmp-1;
                    break;
                case 1:
                    fscanf(file,"%d", &int_tmp);
                    matrix->col_ind[i] = int_tmp-1;
                    break;
                case 2:
                    fscanf(file,"%f", &float_tmp);
                    matrix->values[i] = float_tmp;
                    break;   
            }
        }
    }    
    return matrix;
}

__global__ void matrixVectorMultCuda(int* row_ptr, int* col_ind, float* values, float* x,int row_size , int col_size, int value_size, int *threadsID_ptr_Mapper, int * numberOfIndexesToProcess){
    //printf("ThreadID: %i\n", col_size);
    int tid=threadIdx.x+blockIdx.x*blockDim.x;
    float sum=0;

    //each thread search its matching row id from the list 
    //thread zero is responsible from row 0 - rowsperthreadsin sparse matrix.
    //since the given format for sparse matrix is actually optimized, this for loop 
    //is also runs on O(n) (non-zero value size); 
    if(tid<row_size){
       
        for(int r_i=0; r_i<numberOfIndexesToProcess[tid];r_i++){
            int tid_x = r_i+threadsID_ptr_Mapper[tid]; 
            sum=0;
            //printf("TID[%i]- %i - ",tid,tid_x );
            for(int i = 0; i<value_size; i++){
                if(row_ptr[i]==tid_x){
                    sum += (float) (x[tid_x]*values[i]);
                }
            }
           //printf("[%e]\n",sum );
            __syncthreads();
            x[tid_x]=sum;  
        }
       
          
    }
    //printf("%i\n",tid);
    // All threads will wait till they come to this point
    // We are now quite confident that all array values are updated.
}

__host__ void  outer_VecMatMult_Started(struct SparseMatrix* sparse_Matrix, float *x, int row_size, int col_size, int value_size, 
    int NUMBER_OF_CUDA_THREADS ,int NUMBER_OF_REPETITION , int OUTPUT_FLAG){
    
    int *dev_row_ptr, *dev_col_ind, *dev_threadsID_ptr_Mapper,*dev_numberOfIndexesToProcess;
    float * dev_values, *dev_x;

    int size = sizeof(int)*value_size;
    int size_f = sizeof(float)*value_size;
    int size_f_x = sizeof(float)* row_size; 
  
        
    if(OUTPUT_FLAG==1){
        printf("Initial Matrix:\n");
            printf("    Values Array: [ %e",x[0]);
            for(int i=1; i<value_size;i++){
                printf(", %e",sparse_Matrix->values[i]);
            }
            printf("]\n    Col_Ind Array: [ %i",sparse_Matrix->col_ind[0]);
            for(int i=1; i<value_size;i++){
                printf(", %i",sparse_Matrix->col_ind[i]);
            }
            printf("]\n    Row_Ptr Array: [ %i",sparse_Matrix->row_ptr[0]);
            for(int i=1; i<value_size;i++){
                printf(", %i",sparse_Matrix->row_ptr[i]);
            }
        printf("]\nVector: [ %e",x[0] );
            for(int i=1; i<value_size;i++){
                printf(", %e",x[i]);
        }
        printf("]\n");
    }

    int partitionSize=sizeof(int)*NUMBER_OF_CUDA_THREADS;
    int *threadsID_ptr_Mapper =(int*) malloc(partitionSize);
    int *numberOfIndexesToProcess = (int*) malloc(partitionSize);
    int threadSize,threadSizeRemaining;
    
    if(NUMBER_OF_CUDA_THREADS>row_size){
         threadSize = 1;
        threadSizeRemaining = 0;
         threadsID_ptr_Mapper[0] = 0;
         numberOfIndexesToProcess[0] = threadSize;
         for(int i = 1; i< NUMBER_OF_CUDA_THREADS; i++){
             threadsID_ptr_Mapper[i] =  threadsID_ptr_Mapper[i-1] +  threadSize;            
             numberOfIndexesToProcess[i] = threadSize;         
             //printf("size[%i]: %i - %i\n", i, threadsID_ptr_Mapper[i] ,numberOfIndexesToProcess[i]);     
         }
          
    }
    else{
            threadSize = value_size/NUMBER_OF_CUDA_THREADS;
            threadSizeRemaining = value_size%NUMBER_OF_CUDA_THREADS;
            threadsID_ptr_Mapper[0] = 0;
            numberOfIndexesToProcess[0] = threadSize;
            for(int i = 1; i< NUMBER_OF_CUDA_THREADS; i++){
                if(i<NUMBER_OF_CUDA_THREADS-1){
                    threadsID_ptr_Mapper[i] =  threadsID_ptr_Mapper[i-1] +  threadSize;
                    numberOfIndexesToProcess[i] = threadSize;
                }
                else{
                    threadsID_ptr_Mapper[i] = threadsID_ptr_Mapper[i-1] + threadSize;
                    numberOfIndexesToProcess[i] = threadSize+threadSizeRemaining ;
                }
                //printf("size[%i]: %i - %i\n", i, threadsID_ptr_Mapper[i] ,numberOfIndexesToProcess[i]);    
            }
    }

    

    hipMalloc((void**)&dev_numberOfIndexesToProcess, partitionSize);
    hipMalloc((void**)&dev_threadsID_ptr_Mapper, partitionSize);
    hipMalloc((void**)&dev_row_ptr, size);
    hipMalloc((void**)&dev_col_ind, size);
    hipMalloc((void**)&dev_values, size_f);
    hipMalloc((void**)&dev_x, size_f_x);
    
    hipMemcpy(dev_row_ptr, sparse_Matrix->row_ptr, size,hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ind, sparse_Matrix->col_ind, size,hipMemcpyHostToDevice);
    hipMemcpy(dev_values, sparse_Matrix->values, size,hipMemcpyHostToDevice);
    hipMemcpy(dev_threadsID_ptr_Mapper, threadsID_ptr_Mapper, partitionSize,hipMemcpyHostToDevice);
    hipMemcpy(dev_numberOfIndexesToProcess, numberOfIndexesToProcess, partitionSize,hipMemcpyHostToDevice);
    dim3 threadsPerBlock(1, NUMBER_OF_CUDA_THREADS);
   

    for(int i=0; i<NUMBER_OF_REPETITION; i++){
        hipMemcpy(dev_x, x, size_f_x,hipMemcpyHostToDevice);
        matrixVectorMultCuda<<<1,NUMBER_OF_CUDA_THREADS>>>(dev_row_ptr,dev_col_ind,dev_values,dev_x,row_size,col_size,value_size,dev_threadsID_ptr_Mapper,dev_numberOfIndexesToProcess);
        hipMemcpy(x, dev_x, size_f_x,hipMemcpyDeviceToHost);
        //printf("Iteration %i out of %i Completed\n",i+1,NUMBER_OF_REPETITION);
    }
    if(OUTPUT_FLAG==1||OUTPUT_FLAG==2){
        printf("Resulting Vector: [ %e",x[0] );
        for(int i=1; i<value_size;i++){
            printf(", %e",x[i]);
        }
        printf("]\n");
    }
    hipFree(dev_numberOfIndexesToProcess);
    hipFree(dev_threadsID_ptr_Mapper);
    hipFree(dev_row_ptr);
    hipFree(dev_col_ind);
    hipFree(dev_values);
    hipFree(dev_x);

    free(threadsID_ptr_Mapper);threadsID_ptr_Mapper=NULL;
    free(numberOfIndexesToProcess);numberOfIndexesToProcess=NULL;
}

int main(int argc, char *argv[]){
    clock_t start = clock(), diff;
    int row_size,col_size,value_size;
    
    struct SparseMatrix * sparse_Matrix = readSparseMatrix (argv[4],row_size,col_size,value_size);
    float *x = (float *) malloc(sizeof(float)*row_size);
    for(int i = 0 ; i < row_size ; i++){
        x[i]=1;
    }
    /*printf("%i\n",row_size);
    printf("%i\n",col_size);
    printf("%i\n",value_size);
    printf("%i\n",sparse_Matrix->row_ptr[value_size-2]);
    //Reading Finished*/

    //Cuda zone
    /* initialization */

    int NUMBER_OF_CUDA_THREADS = atoi(argv[1]);
    if(NUMBER_OF_CUDA_THREADS>1024){
        printf("The number of allowed cuda threads is 1024!\nSetting the threads number automatically to 1024.\n------ Resetting the program! ------\n");
        NUMBER_OF_CUDA_THREADS = 1024;
        sleep(1);
    }   
    int NUMBER_OF_REPETITION = atoi(argv[2]);
    int OUTPUT_FLAG = atoi(argv[3]);

    outer_VecMatMult_Started(sparse_Matrix, x, row_size, col_size, value_size, NUMBER_OF_CUDA_THREADS ,NUMBER_OF_REPETITION , OUTPUT_FLAG);
  
    free(sparse_Matrix);
    sparse_Matrix= NULL;

    diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("----\nTime taken: %d seconds %d milliseconds\n", msec/1000, msec%1000);
    return 0;
}

